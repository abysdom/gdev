#include <stdint.h>
#include <hip/hip_runtime.h>
__global__
void add(uint32_t *a, uint32_t *b, uint32_t *c, uint32_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        int idx = i * n + j;
        c[idx] = a[idx] + b[idx];
    }
    else {
        c[0] = 0xdeadbeef;
    }
}
